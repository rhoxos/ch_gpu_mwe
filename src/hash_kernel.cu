#include "hip/hip_runtime.h"
#include "hash_kernel.h"
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <math.h>


using namespace std;

void allocateDeviceMemory(void** M, int size)
{
    hipError_t err = hipMalloc(M, size);
    assert(err==hipSuccess);
}

__global__ void gdata_init(unsigned long long *g_odata, int block_size)
{
    //unsigned int tid = threadIdx.x;
    unsigned int j = blockIdx.x * block_size + threadIdx.x;
    unsigned int thread_size = blockDim.x;
    unsigned int end = (blockIdx.x +1) * block_size;

    //g_odata[tid]=100;
    /*if(blockIdx.x==0)
    {
        printf("My j: %d, My tid: %d\n",j, tid);
    }*/

    while(j<end)
    {
        g_odata[j]=0;
        j+=thread_size;
    }
    //printf("Block Dim: %d // gridDim: %d\n",blockDim.x, gridDim.x);
    //__syncthreads();
}

__global__ void run_hist_both(unsigned long long *g_odata_a, unsigned long long *g_odata_b, int block_size, int data_end)
{
    extern  __shared__  unsigned long long sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int starting_point = blockIdx.x * block_size + threadIdx.x;
    unsigned int j = starting_point;
    unsigned int end = (blockIdx.x +1) * block_size;
    unsigned int thread_size = blockDim.x;

    unsigned int sdata_end =  gridDim.x * thread_size;

    if(tid==0 && blockIdx.x==0)
    {
        printf("Grid Dim (Block Number): %d / Block Dim (Block Size, Thread size): %d /end: %d/ block size parameter: %d/ sdata end: %d\n",gridDim.x,blockDim.x, end, block_size,sdata_end);
    }
    //printf("Starting point: %d\n",starting_point);

    if(end>data_end)
    {
        end=data_end;
    }

    //if(tid==0) printf("Block id: %d / Start: %d / End: %d\n",blockIdx.x,j, sdata_end);

    while(j<sdata_end)///initializing __shared__
    {
        //printf("Current point: %d\n", j);
        sdata[j]=2;
        j+=thread_size;
    }
    __syncthreads();


    //int index = (g_idata_a[j]%gridDim.x)*thread_size+tid;
    //sdata[index]++;//error
    //sdata[0]++;//error
    sdata[0]=1;
    __syncthreads();

    for(int k=0;k<gridDim.x;k++)///recording the first part to g_odata
    {
        //g_odata[k*thread_size+tid] = sdata[k*thread_size+tid];
        //atomicAdd(&g_odata[0], temp);
        atomicAdd(&g_odata_a[k*thread_size+tid], sdata[k*thread_size+tid]);
        //g_odata_a[k*thread_size+tid] = sdata[k*thread_size+tid];
    }
    __syncthreads();

    j=starting_point;
    while(j<sdata_end)///initializing __shared__
    {
        sdata[j]=9;
        j+=thread_size;
    }
    //for(int k=0;k<sdata_end;k++) sdata[k]=0;
    __syncthreads();

    j=starting_point;
    /*while(j<end)
    {
        int index = (g_idata_b[j]%gridDim.x)*thread_size+tid;
        //atomicAdd(&sdata[index], 1);
        j+=thread_size;
    }
    __syncthreads();*/


    for(int k=0;k<gridDim.x;k++)///recording the first the second part to g_odata
    {
        atomicAdd(&g_odata_b[k*thread_size+tid], sdata[k*thread_size+tid]);//adding first_data_part_end so it can have both
        //g_odata_b[k*thread_size+tid] = sdata[k*thread_size+tid];
    }
}


void gpu_main()
{
    //printf("Setting Devices (WIP)\n");


    unsigned long long *GPUO_A;
    unsigned long long *GPUO_B;


    allocateDeviceMemory((void**)&GPUO_A, sizeof(unsigned long long)*(128));
    allocateDeviceMemory((void**)&GPUO_B, sizeof(unsigned long long)*(128));


    int block_size=10;
    int block_no =2;
    int thread_no=4;

    gdata_init<<< block_no, thread_no, sizeof(unsigned long long) * thread_no*block_no >>>((unsigned long long *) GPUO_A, block_size); //each block should look for block_size number of elements
    gdata_init<<< block_no, thread_no, sizeof(unsigned long long) * thread_no*block_no >>>((unsigned long long *) GPUO_B, block_size);

    printf("Init complete\n");

    run_hist_both<<< block_no, thread_no, sizeof(unsigned long long) * thread_no*block_no >>>((unsigned long long *) GPUO_A, (unsigned long long *) GPUO_B, block_size, 128); //each block should look for block_size number of elements
    unsigned long long * gpu_histogram;
    gpu_histogram = new unsigned long long[256];
    hipMemcpy((void **) gpu_histogram, GPUO_A, sizeof(unsigned long long) * 64, hipMemcpyDeviceToHost);
    for(int i=0;i<8;i++)
    {
        printf("A %d: %llu\n",i,gpu_histogram[i]);
    }

    printf("\n\n\n\n\n");

    hipMemcpy((void **) gpu_histogram, GPUO_B, sizeof(unsigned long long) * 64, hipMemcpyDeviceToHost);
    for(int i=0;i<8;i++)
    {
        printf("B %d: %llu\n",i,gpu_histogram[i]);
    }
    return;
}